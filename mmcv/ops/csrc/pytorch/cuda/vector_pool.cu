// Copyright (c) OpenMMLab. All rights reserved.
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "pytorch_cuda_helper.hpp"
#include "vector_pool.cuh"
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

void StackQueryLocalNeighborIdxsCUDAKernelLauncher(
    const Tensor support_xyz_tensor, const Tensor xyz_batch_cnt_tensor,
    const Tensor new_xyz_tensor, const Tensor new_xyz_batch_cnt_tensor,
    Tensor stack_neighbor_idxs_tensor, Tensor start_len_tensor,
    Tensor cumsum_tensor, const int avg_length_of_neighbor_idxs,
    const float max_neighbour_distance, const int nsample,
    const int neighbor_type) {
  int batch_size = xyz_batch_cnt_tensor.size(0);
  int M = new_xyz_tensor.size(0);
  at::cuda::CUDAGuard device_guard(support_xyz_tensor.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // blockIdx.x(col), blockIdx.y(row)
  dim3 blocks(DIVUP(M, THREADS_PER_BLOCK));
  dim3 threads(THREADS_PER_BLOCK);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      support_xyz_tensor.scalar_type(),
      "query_stacked_local_neighbor_idxs_cuda_kernel", [&] {
        query_stacked_local_neighbor_idxs_cuda_kernel<scalar_t>
            <<<blocks, threads, 0, stream>>>(
                support_xyz_tensor.data_ptr<scalar_t>(),
                xyz_batch_cnt_tensor.data_ptr<int>(),
                new_xyz_tensor.data_ptr<scalar_t>(),
                new_xyz_batch_cnt_tensor.data_ptr<int>(),
                stack_neighbor_idxs_tensor.data_ptr<int>(),
                start_len_tensor.data_ptr<int>(), cumsum_tensor.data_ptr<int>(),
                avg_length_of_neighbor_idxs, max_neighbour_distance, batch_size,
                M, nsample, neighbor_type);
      });
  AT_CUDA_CHECK(hipGetLastError());
}

void StackQueryThreeNNLocalIdxsCUDAKernelLauncher(
    const Tensor support_xyz_tensor, const Tensor new_xyz_tensor,
    const Tensor new_xyz_grid_centers_tensor, Tensor new_xyz_grid_idxs_tensor,
    Tensor new_xyz_grid_dist2_tensor, Tensor stack_neighbor_idxs_tensor,
    Tensor start_len_tensor, const int M, const int num_total_grids) {
  at::cuda::CUDAGuard device_guard(support_xyz_tensor.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // blockIdx.x(col), blockIdx.y(row)
  dim3 blocks(DIVUP(M, THREADS_PER_BLOCK));
  dim3 threads(THREADS_PER_BLOCK);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      support_xyz_tensor.scalar_type(),
      "query_three_nn_by_stacked_local_idxs_cuda_kernel", [&] {
        query_three_nn_by_stacked_local_idxs_cuda_kernel<scalar_t>
            <<<blocks, threads, 0, stream>>>(
                support_xyz_tensor.data_ptr<scalar_t>(),
                new_xyz_tensor.data_ptr<scalar_t>(),
                new_xyz_grid_centers_tensor.data_ptr<scalar_t>(),
                new_xyz_grid_idxs_tensor.data_ptr<int>(),
                new_xyz_grid_dist2_tensor.data_ptr<scalar_t>(),
                stack_neighbor_idxs_tensor.data_ptr<int>(),
                start_len_tensor.data_ptr<int>(), M, num_total_grids);
      });
  AT_CUDA_CHECK(hipGetLastError());
}


int StackVectorPoolForwardCUDAKernelLauncher(const Tensor support_xyz_tensor, const Tensor xyz_batch_cnt_tensor,
    const Tensor support_features_tensor, const Tensor new_xyz_tensor, const Tensor new_xyz_batch_cnt_tensor,
    Tensor new_features_tensor, Tensor new_local_xyz_tensor,
    Tensor point_cnt_of_grid_tensor, Tensor grouped_idxs_tensor,
    const int num_grid_x, const int num_grid_y, const int num_grid_z, const float max_neighbour_distance, const int use_xyz,
    const int num_max_sum_points, const int nsample, const int neighbor_type, const int pooling_type){
        at::cuda::CUDAGuard device_guard(support_xyz_tensor.device());
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        const float *support_xyz = support_xyz_tensor.data<float>();
        const float *support_features = support_features_tensor.data<float>();
        const int *xyz_batch_cnt = xyz_batch_cnt_tensor.data<int>();
        const float *new_xyz = new_xyz_tensor.data<float>();
        const int *new_xyz_batch_cnt = new_xyz_batch_cnt_tensor.data<int>();
        float *new_features = new_features_tensor.data<float>();
        float *new_local_xyz = new_local_xyz_tensor.data<float>();
        int *point_cnt_of_grid = point_cnt_of_grid_tensor.data<int>();
        int *grouped_idxs = grouped_idxs_tensor.data<int>();
        int N = support_xyz_tensor.size(0);
        int batch_size = xyz_batch_cnt_tensor.size(0);
        int M = new_xyz_tensor.size(0);
        int num_c_out = new_features_tensor.size(1);
        int num_c_in = support_features_tensor.size(1);
        int num_total_grids = point_cnt_of_grid_tensor.size(1);

        int num_c_each_grid = num_c_out / num_total_grids;
        float grid_size_x = max_neighbour_distance * 2 / num_grid_x;
        float grid_size_y = max_neighbour_distance * 2 / num_grid_y;
        float grid_size_z = max_neighbour_distance * 2 / num_grid_z;
        dim3 blocks(DIVUP(M, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
        dim3 threads(THREADS_PER_BLOCK);
        int cum_sum = 0;
        printf("%d\n",cum_sum);
        int *p_cum_sum;
        hipMalloc((void**)&p_cum_sum, sizeof(int));
        hipMemcpy(p_cum_sum, &cum_sum, sizeof(int), hipMemcpyHostToDevice);
        stack_vector_pool_cuda_kernel<<<blocks, threads>>>(
            support_xyz, support_features, xyz_batch_cnt,
            new_xyz, new_features, new_local_xyz, new_xyz_batch_cnt,
            num_grid_x, num_grid_y, num_grid_z, max_neighbour_distance,
            batch_size, M, num_c_in, num_c_out,
            num_c_each_grid, num_total_grids, point_cnt_of_grid, grouped_idxs,
            use_xyz, grid_size_x, grid_size_y, grid_size_z, p_cum_sum, num_max_sum_points,
            nsample, neighbor_type, pooling_type
        );

        hipMemcpy(&cum_sum, p_cum_sum, sizeof(int), hipMemcpyDeviceToHost);
                printf("%d\n",cum_sum);
        AT_CUDA_CHECK(hipGetLastError());
        return cum_sum;
    }

void StackVectorPoolBackwardCUDAKernelLauncher(const Tensor grad_new_features_tensor,
    const Tensor point_cnt_of_grid_tensor, const Tensor grouped_idxs_tensor,
    Tensor grad_support_features_tensor){
        at::cuda::CUDAGuard device_guard(grad_new_features_tensor.device());
        hipStream_t stream = at::cuda::getCurrentCUDAStream();
        int M = grad_new_features_tensor.size(0);
        int num_c_out = grad_new_features_tensor.size(1);
        int N = grad_support_features_tensor.size(0);
        int num_c_in = grad_support_features_tensor.size(1);
        int num_total_grids = point_cnt_of_grid_tensor.size(1);
        int num_max_sum_points = grouped_idxs_tensor.size(0);

        const float *grad_new_features = grad_new_features_tensor.data<float>();
        const int *point_cnt_of_grid = point_cnt_of_grid_tensor.data<int>();
        const int *grouped_idxs = grouped_idxs_tensor.data<int>();
        float *grad_support_features = grad_support_features_tensor.data<float>();

        int num_c_each_grid = num_c_out / num_total_grids;
        dim3 blocks(DIVUP(num_max_sum_points, THREADS_PER_BLOCK), num_c_in);  // blockIdx.x(col), blockIdx.y(row)
        dim3 threads(THREADS_PER_BLOCK);
        stack_vector_pool_backward_cuda_kernel<<<blocks, threads>>>(
            grad_new_features, point_cnt_of_grid, grouped_idxs, grad_support_features,
            N, M, num_c_out, num_c_in, num_c_each_grid, num_total_grids, num_max_sum_points
        );
        AT_CUDA_CHECK(hipGetLastError());
    }